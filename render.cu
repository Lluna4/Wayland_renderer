#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "game_lib/library.h"

__global__ void set_pixel(unsigned char *pixel, int r, int g, int b, int width)
{
    int x  = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    int pxl_index = (x + width * y) * 4;
    pixel[pxl_index] = r;
    pxl_index++;
    pixel[pxl_index] = g;
    pxl_index++;
    pixel[pxl_index] = b;
    pxl_index++;
    pixel[pxl_index] = 255;
}

__global__ void set_pixel_offset(unsigned char *pixel, int r, int g, int b, int width, int offset_x, int offset_y)
{
    int x  = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
    int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

    //printf("%i, %i\n", offset_x, offset_y);
    int pxl_index = (x + width * y) * 4;
    pixel[pxl_index] = r;
    pxl_index++;
    pixel[pxl_index] = g;
    pxl_index++;
    pixel[pxl_index] = b;
    pxl_index++;
    pixel[pxl_index] = 255;
}



extern "C" void prepare_pixels(unsigned char *pxl, int width, int height)
{
    unsigned char *pxl2;
    hipMalloc(&pxl2, ((width * height) * 4) * sizeof(unsigned char));
    hipMemcpy(pxl2, pxl, ((width * height) * 4), hipMemcpyHostToDevice);

    set_pixel<<<height, width>>>(pxl2, 0, 0, 0, 2);

    hipDeviceSynchronize();
    hipMemcpy(pxl, pxl2, ((width * height) * 4), hipMemcpyDeviceToHost);
    hipFree(pxl2);
}

extern "C" void render_shapes(unsigned char *pxl, int width, int height, geometrical_4axis *shapes, int shapes_size)
{
    unsigned char *pxl2;
    hipMalloc(&pxl2, ((width * height) * 4) * sizeof(unsigned char));
    hipMemcpy(pxl2, pxl, ((width * height) * 4), hipMemcpyHostToDevice);

    for (int i = 0; i < shapes_size; i++)
    {
        int width_shape = shapes[i].x_end - shapes[i].x_start;
        int height_shape = shapes[i].y_end - shapes[i].y_start;
        color_alpha col = shapes[i].color;
        const dim3 blockSize(8, 8);
        const dim3 gridSize(width_shape/blockSize.x, height_shape/blockSize.y); 
        set_pixel_offset<<<blockSize, gridSize>>>(pxl2, col.r, col.g, col.b, width, shapes[i].x_start, shapes[i].y_start);
        hipDeviceSynchronize();
    }
    hipMemcpy(pxl, pxl2, ((width * height) * 4), hipMemcpyDeviceToHost);
    hipFree(pxl2);
}

extern "C" int number_dev()
{
    int ret;
    hipGetDeviceCount(&ret);

    return ret;
}